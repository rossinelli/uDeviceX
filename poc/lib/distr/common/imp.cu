#include "hip/hip_runtime.h"
#include <conf.h>
#include "inc/conf.h"

#include "d/api.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "utils/kl.h"
#include "utils/cc.h"
#include "frag/dev.h"
#include "frag/imp.h"

#include "distr/map/type.h"

#include "imp.h"

namespace dcommon_dev {
#include "dev.h"
}

void dcommon_pack_pp_packets(int nc, int nv, const Particle *pp, DMap m, /**/ Sarray<Particle*, 27> buf) {
    enum {THR=128};
    dim3 thrd(THR, 1);
    dim3 blck(ceiln(nv, THR), nc);

    KL(dcommon_dev::pack_pp_packets, (blck, thrd), (nv, pp, m, /**/ buf));
}

void dcommon_shift_one_frag(int3 L, int n, const int fid, /**/ Particle *pp) {
    KL(dcommon_dev::shift_one_frag, (k_cnf(n)), (L, n, fid, /**/ pp));
}

void dcommon_shift_halo(int3 L, int nhalo, const Sarray<int, 27> starts, /**/ Particle *pp) {
    KL(dcommon_dev::shift_halo, (k_cnf(nhalo)), (L, starts, /**/ pp));
}
