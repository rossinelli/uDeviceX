#include <stdio.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"

#include "inc/def.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "math/dev.h"
#include "math/tri/dev.h"

#include "io/mesh_read/imp.h"

#include "utils/error.h"
#include "utils/msg.h"
#include "utils/imp.h"

#include "d/q.h"
#include "d/api.h"
#include "d/ker.h"
#include "utils/cc.h"
#include "utils/kl.h"

#include "rbc/params/imp.h"
#include "rbc/adj/imp.h"
#include "rbc/shape/imp.h"
#include "bending/imp.h"

#include "rnd/imp.h"
#include "rbc/type.h"

#include "rbc/adj/type/common.h"
#include "rbc/adj/type/dev.h"
#include "rbc/adj/dev.h"
#include "area_volume/imp.h"

#include "imp.h"
#include "imp/type.h"

namespace rbc_force_dev {
#include "dev/type.h"
#include "dev/fetch.h"
#include "dev/common.h"
#include "dev/main.h"
}

#include "imp/main.h"
#include "imp/forces.h"
#include "imp/stat.h"
