#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <conf.h>
#include "inc/conf.h"
#include "inc/def.h"
#include "utils/error.h"
#include "utils/msg.h"
#include "utils/cc.h"

#include "d/q.h"
#include "d/ker.h"
#include "d/api.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "utils/texo.h"
#include "utils/texo.dev.h"
#include "mesh/triangles/type.h"
#include "mesh/triangles/imp.h"

#include "utils/kl.h"
#include "rigid/imp.h"

#include "imp.h"

enum {X, Y, Z};

static __host__ __device__ bool same_side(const float *x, const float *p, const float *a, const float *b, const float *inplane) {
    const float n[3] = {a[Y] * b[Z] - a[Z] * b[Y],
                        a[Z] * b[X] - a[X] * b[Z],
                        a[X] * b[Y] - a[Y] * b[X]};

    const float ndx = n[X] * (x[X] - inplane[X]) + n[Y] * (x[Y] - inplane[Y]) + n[Z] * (x[Z] - inplane[Z]);
    const float ndp = n[X] * (p[X] - inplane[X]) + n[Y] * (p[Y] - inplane[Y]) + n[Z] * (p[Z] - inplane[Z]);

    return ndx * ndp > 0;
}

static __host__ __device__ bool in_tetrahedron(const float *x, const float *A, const float *B, const float *C, const float *D) {
    const float AB[3] = {B[X] - A[X], B[Y] - A[Y], B[Z] - A[Z]};
    const float AC[3] = {C[X] - A[X], C[Y] - A[Y], C[Z] - A[Z]};
    const float AD[3] = {D[X] - A[X], D[Y] - A[Y], D[Z] - A[Z]};

    const float BC[3] = {C[X] - B[X], C[Y] - B[Y], C[Z] - B[Z]};
    const float BD[3] = {D[X] - B[X], D[Y] - B[Y], D[Z] - B[Z]};

    return
        same_side(x, A, BC, BD, D) &&
        same_side(x, B, AD, AC, D) &&
        same_side(x, C, AB, BD, D) &&
        same_side(x, D, AB, AC, A);
}

int collision_inside_1p(int spdir, const float *r, const float *vv, const int4 *tt, const int nt) {
    int c = 0;
    float origin[3] = {0, 0, 0};

    if (spdir != NOT_PERIODIC)
        origin[spdir] = r[spdir];

    for (int i = 0; i < nt; ++i) {
        int4 t = tt[i];
        if (in_tetrahedron(r, vv + 3*t.x, vv + 3*t.y, vv + 3*t.z, origin)) ++c;
    }
    return c%2;
}

namespace collision_dev
{
__global__ void init_tags(const int n, const int color, /**/ int *tags) {
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < n) tags[gid] = color;
}

union Pos {
    float2 f2[2];
    struct { float r[3]; float dummy; };
};

static __device__ Pos fetchPos(const Particle *vert, const int id) {
    const float2 *pp = (const float2*) vert;
    Pos r;
    r.f2[0] = pp[3 * id + 0];
    r.f2[1] = pp[3 * id + 1];
    return r;
}

static __device__ bool inside_box(const float r[3], float3 lo, float3 hi) {
    enum {X, Y, Z};
    return
        r[X] >= lo.x && r[X] <= hi.x &&
        r[Y] >= lo.y && r[Y] <= hi.y &&
        r[Z] >= lo.z && r[Z] <= hi.z;
}

/* assume nm blocks along y */
__global__ void label(int pdir, const Particle *pp, const int n, const Particle *vert, const int nv,
                      Triangles tri, const float3 *minext, const float3 *maxext,
                      int lab_in, /**/ int *labels) {
    int i, sid, gid, count, mbase;
    Particle p;
    Pos a, b, c;
    float3 lo, hi;
    int4 t;
    sid = blockIdx.y;
    gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;

    count = 0;

    p = pp[gid];
    
    lo = minext[sid];
    hi = maxext[sid];
    if (!inside_box(p.r, lo, hi)) return;

    float origin[3] = {0, 0, 0};
    if (pdir != NOT_PERIODIC) origin[pdir] = p.r[pdir];

    mbase = nv * sid;
    for (i = 0; i < tri.nt; ++i) {
        t = tri.tt[i];

        a = fetchPos(vert, mbase + t.x);
        b = fetchPos(vert, mbase + t.y);
        c = fetchPos(vert, mbase + t.z);

        if (in_tetrahedron(p.r, a.r, b.r, c.r, origin)) ++count;
    }

    // dont consider the case of inside several solids
    if (count % 2) atomicExch(labels + gid, lab_in);
}
}

void collision_label_ini(long n, int lab_out, /**/ int *labels) {
    KL(collision_dev::init_tags, (k_cnf(n)), (n, lab_out, /**/ labels));
}

void collision_label(int pdir, int n, const Particle *pp, const Triangles *tri, 
                     int nv, int nm, const Particle *i_pp, 
                     const float3 *minext, const float3 *maxext,
                     int lab_in, /**/ int *labels) {
    enum {THR = 128};
    
    dim3 thrd(THR, 1);
    dim3 blck(ceiln(n, THR), nm);

    if (nm == 0 || n == 0) return;    
    
    KL(collision_dev::label, (blck, thrd),
       (pdir, pp, n, i_pp, nv, *tri, minext, maxext, lab_in, /**/ labels)); 

}
