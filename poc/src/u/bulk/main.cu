#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <string.h>
#include <conf.h>
#include "inc/conf.h"

#include "d/api.h"
#include "utils/msg.h"
#include "utils/error.h"
#include "utils/imp.h"
#include "utils/cc.h"
#include "utils/mc.h"

#include "mpi/glb.h"
#include "mpi/wrapper.h"
#include "inc/dev.h"
#include "inc/type.h"
#include "conf/imp.h"
#include "struct/partlist/type.h"
#include "clist/imp.h"
#include "pair/imp.h"

#include "coords/ini.h"
#include "coords/imp.h"

#include "struct/parray/imp.h"
#include "struct/farray/imp.h"
#include "flu/type.h"
#include "fluforces/imp.h"

#include "io/txt/imp.h"

static Particle *pp, *pp0, *pp_hst;
static Force *ff, *ff_hst;
static int n;
static Clist *clist;
static ClistMap *cmap;
static FluForcesBulk *bulkforces;

static void read_pp(const char *fname) {
    TxtRead *tr;
    long sz;
    UC(txt_read_pp(fname, &tr));
    n = txt_read_get_n(tr);
    msg_print("have read %d particles", n);

    sz = n + 32;

    EMALLOC(sz, &pp_hst);
    EMALLOC(sz, &ff_hst);

    Dalloc(&pp,  sz);
    Dalloc(&pp0, sz);
    Dalloc(&ff,  sz);

    memcpy(pp_hst, txt_read_get_pp(tr), sz * sizeof(Particle));
    CC(d::Memcpy(pp, pp_hst, sz * sizeof(Particle), H2D));
    CC(d::Memset(ff, 0, sz * sizeof(Force)));

    UC(txt_read_fin(tr));
}

static void dealloc() {
    Dfree(pp);
    Dfree(pp0);
    Dfree(ff);
    EFREE(pp_hst);
    EFREE(ff_hst);
    n = 0;
}

static void build_clist() {
    UC(clist_build(n, n, pp, /**/ pp0, clist, cmap));
    Particle *tmp = pp;
    pp = pp0;
    pp0 = tmp;
}

static void set_params(const Config *cfg, float dt, PairParams *p) {
    float kBT;
    UC(conf_lookup_float(cfg, "glb.kBT", &kBT));
    msg_print("kBT: %g", kBT);
    UC(pair_set_conf(cfg, "flu", p));
    UC(pair_compute_dpd_sigma(kBT, dt, /**/ p));
}

int main(int argc, char **argv) {
    Config *cfg;
    const char *fin, *fout;
    PaArray parray;
    FoArray farray;
    Coords *coords;
    int maxp;
    int3 L;
    PairParams *params;
    float dt;
    int rank, dims[3];
    MPI_Comm cart;
    
    m::ini(&argc, &argv);
    m::get_dims(&argc, &argv, dims);
    m::get_cart(MPI_COMM_WORLD, dims, &cart);
    
    MC(m::Comm_rank(cart, &rank));
    msg_ini(rank);

    UC(conf_ini(&cfg));
    UC(conf_read(argc, argv, cfg));

    UC(coords_ini_conf(cart, cfg, &coords));
    L = subdomain(coords);

    UC(pair_ini(&params));
    UC(conf_lookup_float(cfg, "time.dt", &dt));
    UC(set_params(cfg, dt, params));

    UC(conf_lookup_string(cfg, "in", &fin));
    UC(conf_lookup_string(cfg, "out", &fout));
    UC(read_pp(fin));

    maxp = n + 32;

    UC(clist_ini(L.x, L.y, L.z, &clist));
    UC(clist_ini_map(maxp, 1, clist, &cmap));
    UC(build_clist());

    UC(fluforces_bulk_ini(L, maxp, &bulkforces));

    parray_push_pp(pp, &parray);
    farray_push_ff(ff, &farray);

    UC(fluforces_bulk_prepare(n, &parray, /**/ bulkforces));
    UC(fluforces_bulk_apply(params, n, bulkforces, clists_get_ss(clist), clists_get_cc(clist), /**/ &farray));

    // particles are reordered because of clists
    CC(d::Memcpy(pp_hst, pp, n*sizeof(Particle), D2H));
    CC(d::Memcpy(ff_hst, ff, n*sizeof(Force)   , D2H));
    UC(txt_write_pp_ff(n, pp_hst, ff_hst, fout));

    UC(fluforces_bulk_fin(bulkforces));
    UC(clist_fin(clist));
    UC(clist_fin_map(cmap));
    UC(dealloc());

    UC(pair_fin(params));
    UC(coords_fin(coords));
    UC(conf_fin(cfg));

    MC(m::Barrier(cart));
    m::fin();
}
