#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>

#include <conf.h>
#include "inc/conf.h"

#include "utils/msg.h"
#include "mpi/wrapper.h" /* mini-MPI and -device */
#include "mpi/glb.h"

#include "d/api.h"

#include "utils/error.h"
#include "utils/cc.h"
#include "utils/mc.h"
#include "utils/kl.h"
#include "conf/imp.h"
#include "inc/type.h"
#include "inc/dev.h"
#include "dbg/imp.h"
#include "coords/ini.h"
#include "coords/imp.h"

const int n = 10;
Particle *pp;
Force *ff;

void alloc() {
    Dalloc(&pp, n);
    Dalloc(&ff, n);
}

void free() {
    Dfree(pp);
    Dfree(ff);
}

namespace dev {

__global__ void fill_bugs(int3 L, Particle *pp, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    Particle p;
    p.r[0] = p.r[1] = p.r[2] = 0;
    p.v[0] = p.v[1] = p.v[2] = 0;

    if (i >= n) return;
    if (i == 1) p.r[0] = 1.5 * L.x;  // invalid position
    if (i <  1) p.v[0] = 0.f / 0.f; // nan
    pp[i] = p;
}

__global__ void fill_bugs(Force *ff, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    Force f;
    f.f[0] = f.f[1] = f.f[2] = 0;

    if (i >= n) return;
    if (i < 1) f.f[0] = 1.f / 0.f; // inf
    ff[i] = f;
}
} // dev

void fill_bugs(int3 L) {
    KL(dev::fill_bugs, (k_cnf(n)), (L, pp, n));
    KL(dev::fill_bugs, (k_cnf(n)), (ff, n));
}

void check(float dt, const Coords *c, Dbg *dbg) {
    UC(dbg_check_pos    (c, "flu", dbg, n, pp));
    UC(dbg_check_vel    (dt, c, "flu", dbg, n, pp));
    UC(dbg_check_forces (dt, c, "flu.ff", dbg, n, pp, ff));
}

int main(int argc, char **argv) {
    Dbg *dbg;
    Config *cfg;
    Coords *coords;
    int3 L;
    float dt;
    int dims[3];
    MPI_Comm cart;

    m::ini(&argc, &argv);
    m::get_dims(&argc, &argv, dims);
    m::get_cart(MPI_COMM_WORLD, dims, &cart);

    UC(conf_ini(&cfg));
    UC(dbg_ini(&dbg));
    UC(conf_read(argc, argv, cfg));
    UC(conf_lookup_float(cfg, "time.dt", &dt));
    UC(dbg_set_conf(cfg, dbg));
    UC(coords_ini_conf(cart, cfg, &coords));

    L = subdomain(coords);

    alloc();
    fill_bugs(L);
    check(dt, coords, dbg);
    free();
    UC(dbg_fin(dbg));
    UC(conf_fin(cfg));
    UC(coords_fin(coords));
    MC(m::Barrier(cart));
    m::fin();
}
