#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <mpi.h>

#include <conf.h>
#include "inc/conf.h"

#include "d/ker.h"
#include "d/api.h"
#include "utils/msg.h"

#include "mpi/glb.h"
#include "mpi/wrapper.h"

#include "inc/type.h"
#include "inc/dev.h"
#include "utils/cc.h"
#include "utils/kl.h"
#include "utils/mc.h"
#include "utils/error.h" 
#include "coords/type.h"
#include "coords/ini.h"
#include "coords/imp.h"
#include "conf/imp.h"
#include "wall/wvel/type.h"

#include "wall/sdf/imp.h"
#include "math/tform/type.h"
#include "math/tform/dev.h"

#include "wall/sdf/tex3d/type.h"
#include "wall/sdf/type.h"

#include "wall/sdf/dev.h"
#include "wall/sdf/imp/type.h"

namespace dev {
#include "dev.h"
}

#define DUMP true
#define NO_DUMP false

struct Part { float x, y, z; };

void main0(Sdf *sdf, Part *p) {
    Sdf_v sdf_v;
    float x, y, z;
    x = p->x; y = p->y; z = p->z;
    sdf_get_view(sdf, &sdf_v);
    KL(dev::main, (1, 1), (sdf_v, x, y, z));
}

void main1(MPI_Comm cart, const Coords *c, Part *p) {
    Sdf *sdf;
    int3 L;
    L = subdomain(c);
    UC(sdf_ini(L, &sdf));
    UC(sdf_gen(c, cart, DUMP, sdf));
    UC(main0(sdf, p));
    UC(sdf_fin(sdf));
    dSync();    
}

void read_part(const Config *cfg, /**/ Part *p) {
    enum {X, Y, Z, D};
    float r[D];
    int n;

    UC(conf_lookup_vfloat(cfg, "pos", D, &n, r));
    
    p->x = r[X];
    p->y = r[Y];
    p->z = r[Z];
}

int main(int argc, char **argv) {
    Part p;
    Coords *coords;
    Config *cfg;    
    MPI_Comm cart;
    int dims[3];

    m::ini(&argc, &argv);
    m::get_dims(&argc, &argv, dims);
    m::get_cart(MPI_COMM_WORLD, dims, &cart);

    UC(conf_ini(&cfg));
    UC(conf_read(argc, argv, cfg));
    UC(read_part(cfg, /**/ &p));
    UC(coords_ini_conf(cart, cfg, &coords));

    UC(main1(cart, coords, &p));

    UC(conf_fin(cfg));
    UC(coords_fin(coords));

    MC(m::Barrier(cart));
    m::fin();
}
