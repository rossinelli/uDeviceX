#include <stdio.h>
#include <assert.h>
#include <mpi.h>

#include <conf.h>
#include "inc/conf.h"

#include "utils/error.h"
#include "utils/imp.h"

#include "d/api.h"
#include "utils/msg.h"

#include "inc/def.h"
#include "inc/type.h"
#include "inc/dev.h"

#include "conf/imp.h"

#include "utils/texo.h"
#include "utils/cc.h"

#include "coords/type.h"

#include "rbc/params/imp.h"
#include "rbc/type.h"
#include "rbc/force/area_volume/imp.h"
#include "rbc/imp.h"
#include "rbc/force/rnd/imp.h"
#include "rbc/force/imp.h"
#include "rbc/stretch/imp.h"

#include "scheme/move/imp.h"
#include "scheme/force/imp.h"
#include "scheme/time_line/imp.h" 

#include "io/mesh/imp.h"
#include "io/diag/part/imp.h"
#include "io/mesh_read/imp.h"

#include "mpi/wrapper.h"
#include "mpi/glb.h"

#include "imp.h"

#include "imp/main.h"
