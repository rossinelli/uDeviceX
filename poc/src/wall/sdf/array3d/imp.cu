#include <stdio.h>

#include "conf.h"
#include "inc/conf.h"

#include "d/api.h"

#include "utils/imp.h"
#include "utils/error.h"
#include "utils/cc.h"
#include "utils/msg.h"

#include "type.h"
#include "imp.h"

void array3d_ini(Array3d **pq, size_t x, size_t y, size_t z) {
    Array3d *q;
    hipChannelFormatDesc fmt;
    EMALLOC(1, &q);

    fmt = hipCreateChannelDesc<float>();
    msg_print("alloc cuda 3D Array: %ld %ld %ld", x, y, z);
    CC(hipMalloc3DArray(&q->a, &fmt, make_hipExtent(x, y, z)));
    q->x = x; q->y = y; q->z = z;

    *pq = q;
}

void array3d_fin(Array3d *q) {
    CC(hipFreeArray(q->a));
    EFREE(q);
}

static int good(size_t x, size_t y, size_t z, Array3d *q) {
    return x == q->x && y == q->y && z == q->z;
}

void array3d_copy(size_t x, size_t y, size_t z, float *D, /**/ Array3d *q) {
    hipMemcpy3DParms copyParams;
    if (!good(x, y, z, q))
        ERR("wrong size: %ld, %ld, %ld   !=   %ld, %ld, %ld",
            x, y, z, q->x, q->y, q->z);
    memset(&copyParams, 0, sizeof(copyParams));
    copyParams.srcPtr = make_hipPitchedPtr((void*)D, x*sizeof(float), x, y);
    copyParams.dstArray = q->a;
    copyParams.extent = make_hipExtent(x, y, z);
    copyParams.kind = hipMemcpyHostToDevice;
    CC(hipMemcpy3D(&copyParams));
}
